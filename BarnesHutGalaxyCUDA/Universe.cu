#include "hip/hip_runtime.h"
#include "Universe.h"

#include <ctime>

// TO REMOVE
#include <iostream>

#include "glm/glm.hpp"
#include "glm/gtx/transform.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include ""

float float_rand(float min, float max)
{
	float scale = rand() / (float)RAND_MAX; /* [0, 1.0] */
	return min + scale * (max - min);      /* [min, max] */
}

//======================================
// Kernels
__device__ float3 sub(float3 a, float3 b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ float length(float3 a)
{
	return sqrt((a.x*a.x)+(a.y*a.y)+(a.z*a.z));
}
__device__ float3 add(float3 a, float3 b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void calcForce(int numberOfStars, const Star* inStars, Star* outStars, float deltaTime)
{
	// Equivalent index in the array based on block size and thread
	const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	// Star that will be compared in this CUDA thread
	const Star& inputStar = inStars[index];

	float3 inPos = make_float3(inputStar.position.x, inputStar.position.y, 0.0f);

	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
	for (int i = 0; i < numberOfStars; i++)
	{
		if (i != index)
		{
			const Star& currStar = inStars[i];
			float3 curPos = make_float3(currStar.position.x, currStar.position.y, 0.0f);

			float3 difference = sub(curPos, inPos);
			float distance = length(difference) + SOFTENER;

			if (distance > 0)
			{
				float g = GRAVITATIONAL_CONSTANT * PARTICLE_MASS / (distance*distance*distance);
				acc.x += g * difference.x;
				acc.y += g * difference.y;
			}
		}
	}
	Star& outputStar = outStars[index];

	float3 starVel = make_float3(inputStar.velocity.x, inputStar.velocity.y, 0.0f);
	float3 outVel = add(starVel, make_float3(acc.x*deltaTime, acc.y*deltaTime, 0.0f));
	float3 outPos = add(inPos, make_float3(outVel.x*deltaTime, outVel.y*deltaTime, 0.0f));
	
	outputStar.position.x = outPos.x;
	outputStar.position.y = outPos.y;
	outputStar.velocity.x = outVel.x;
	outputStar.velocity.y = outVel.y;
}
//======================================

Universe::Universe( GLuint numberOfGalaxies, GLfloat galaxyRadius, GLuint numberOfStars )
	: _numberOfGalaxies(numberOfGalaxies)
	, _numberOfStars(numberOfStars)
{
	srand(time(NULL));

	_totalStars = _numberOfGalaxies * _numberOfStars;

	_galaxies.resize(_numberOfGalaxies);

	// Generating universe
	for (auto& galaxy : _galaxies)
	{

		galaxy.radius = galaxyRadius;
		galaxy.center = {float_rand(-1,1), float_rand(-1,1), 0.0f};
		//galaxy.center = { 0.0f,0.0f,0.0f };

		_stars.reserve(_numberOfStars);
		for (GLuint i = 0; i < _numberOfStars; i++)
		{
			float a = float_rand(0, 1) * 2 * M_PI;
			float r = galaxy.radius * sqrt(float_rand(0, 1));

			float displacementX = r * cos(a);
			float displacementY = r * sin(a);

			glm::mat4 spinMat(1.0f);
			glm::vec4 spinVec(displacementX, displacementY, 0.0f, 0.0f);
			spinMat = glm::rotate(spinMat, glm::radians(-90.0f), { 0.0f, 0.0f, 1.0f });
			spinVec = INITIAL_SPIN_FACTOR * (spinMat * spinVec);

			_stars.push_back({
				PARTICLE_MASS,
				{ galaxy.center.x + displacementX, galaxy.center.y + displacementY, 0.0f },
				spinVec,
				{0.0f,0.0f,0.0f}
				});
		}

#ifdef ADD_BLACK_HOLE
		_stars.push_back({ BLACK_HOLE_MASS,{ galaxy.center.x, galaxy.center.y, 0.0f } });
#endif
	}

	// Device prep
	hipMalloc(&_dinStars, _totalStars * sizeof(Star));
	hipMalloc(&_doutStars, _totalStars * sizeof(Star));

	hipMemcpy(_dinStars, _stars.data(), _totalStars * sizeof(Star), hipMemcpyHostToDevice);
	hipMemcpy(_doutStars, _stars.data(), _totalStars * sizeof(Star), hipMemcpyHostToDevice);
}

void Universe::step(float deltaTime)
{
	calcForce << < ceil(_totalStars /(float)BLOCKSIZE), BLOCKSIZE >> >(_totalStars, _dinStars, _doutStars, deltaTime);

	hipMemcpy(_stars.data(), _doutStars, _totalStars * sizeof(Star), hipMemcpyDeviceToHost);
	hipMemcpy(_dinStars, _doutStars, _totalStars * sizeof(Star), hipMemcpyDeviceToDevice);
}

std::vector<glm::vec3> Universe::getVertices() const
{
	std::vector<glm::vec3> verts;
	verts.reserve(_numberOfStars);
	for (auto& star : _stars)
	{
		verts.push_back(star.position);
	}
	return verts;
}

Universe::~Universe()
{
	hipFree(_dinStars);
	hipFree(_doutStars);
}
