#include "hip/hip_runtime.h"
#include "cuda_computes.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <cmath>

__global__ void computeForce(Star* stars, Node* tree, float cellRadius, int numberOfStars)
{
	// One stack per thread block
	__shared__ float depth[(MAXDEPTH * BLOCKSIZE) / WARPSIZE];
	__shared__ int stack[(MAXDEPTH * BLOCKSIZE) / WARPSIZE];

	// Equivalent index in the array based on block size and current work thread
	const int warpGroupIdx = threadIdx.x / warpSize;
	const int warpIdx = threadIdx.x % warpSize;
	const int stride = blockDim.x * gridDim.x;
	const int stackStartIdx = MAXDEPTH * warpGroupIdx;
	int starIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Getting the number of valid children from the root node
	int stackOffset = -1;
	for (int i = 0; i < 4; i++)
	{
		if (tree[0].children[i] != -1)
		{
			stackOffset++;
		}
	}

	// Compute acceleration for every star in the block
	while (starIdx < numberOfStars)
	{
		float3 pos = stars[starIdx].position;
		float3 acc = make_float3(0.0f);

		// Initialize the stack since we are on the first thread on the warp
		if (warpIdx == 0)
		{
			for (int i = 0; i < 4; i++)
			{
				int childIdx = 0;
				if (tree[0].children[i] != -1)
				{
					stack[stackStartIdx + childIdx] = tree[0].children[i];
					depth[stackStartIdx + childIdx] = cellRadius * cellRadius / THETA;
					childIdx++;
				}
			}
		}
		// Making sure all threads on the warp have the updated stack
		__syncthreads();

		// While this warp's stack is not empty, perform the acceleration calculations
		int top = stackOffset + stackStartIdx;
		while (top >= stackStartIdx)
		{
			int nodeIdx = stack[top];
			float nextQuadrantSize = 0.25*depth[top];
			for (int i = 0; i < 4; i++)
			{
				int childIdx = tree[nodeIdx].children[i];

				if (childIdx > -1)
				{
					float3 difference = tree[childIdx].centerOfMass - pos;
					double squaredDist = dot(difference, difference) + SOFTENER;

					// Compute acceleration if leaf node or meets the criterion
					if (tree[childIdx].starIndex != -1 || __all(nextQuadrantSize <= squaredDist))
					{
						double inverseDist = rsqrtf(squaredDist);
						float g = tree[childIdx].mass * inverseDist * inverseDist * inverseDist * GRAVITATIONAL_CONSTANT;
						acc += g * difference;
					}
					else
					{
						if (warpIdx == 0)
						{
							stack[top] = childIdx;
							depth[top] = nextQuadrantSize;
						}
						top++;
					}
				}
			}
			top--;
		}
		// Updating the star's acceleration
		stars[starIdx].acceleration = acc;

		// Jumping one grid to make sure we're not working on the same star as another thread
		starIdx += stride;

		// Make sure that the next iteration of the loop has updated values
		__syncthreads();
	}
}

__global__ void integrate(Star* stars, int numberOfStars, float deltaTime)
{
	const int starIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (starIdx < numberOfStars)
	{
		stars[starIdx].velocity += stars[starIdx].acceleration * deltaTime;
		stars[starIdx].position += stars[starIdx].velocity * deltaTime;
	}
}

//__global__ void calcForce(int numberOfStars, const Star* inStars, Star* outStars, float deltaTime)
//{
//	// Equivalent index in the array based on block size and thread
//	const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
//
//	// Star that will be compared in this CUDA thread
//	const Star& inputStar = inStars[index];
//
//	float3 inPos = make_float3(inputStar.position.x, inputStar.position.y, 0.0f);
//
//	float3 acc = make_float3(0.0f, 0.0f, 0.0f);
//	for (int i = 0; i < numberOfStars; i++)
//	{
//		if (i != index)
//		{
//			const Star& currStar = inStars[i];
//			float3 curPos = make_float3(currStar.position.x, currStar.position.y, 0.0f);
//
//			float3 difference = sub(curPos, inPos);
//			float distance = length(difference) + SOFTENER;
//
//			if (distance > 0)
//			{
//				float g = GRAVITATIONAL_CONSTANT * PARTICLE_MASS / (distance*distance*distance);
//				acc.x += g * difference.x;
//				acc.y += g * difference.y;
//			}
//		}
//	}
//	Star& outputStar = outStars[index];
//
//	float3 starVel = make_float3(inputStar.velocity.x, inputStar.velocity.y, 0.0f);
//	float3 outVel = add(starVel, make_float3(acc.x*deltaTime, acc.y*deltaTime, 0.0f));
//	float3 outPos = add(inPos, make_float3(outVel.x*deltaTime, outVel.y*deltaTime, 0.0f));
//
//	outputStar.position.x = outPos.x;
//	outputStar.position.y = outPos.y;
//	outputStar.velocity.x = outVel.x;
//	outputStar.velocity.y = outVel.y;
//}